#include "hip/hip_runtime.h"

//#include "projects/stereo_traversability_experiments/aras/libstereo_test/gpu_finroc_test/gpu_part.h"
#include "gpu_part.h"

namespace finroc
{
namespace stereo_traversability_experiments
{
namespace aras
{
namespace libstereo_test
{
namespace gpu
{

gpu_part::gpu_part() {}

gpu_part::~gpu_part() {}

//************************************
void gpu_part::GPU_preProcess(uchar **h_inputImageGray_1, uchar **h_inputImageGray_2, uchar **h_outputImageGray,
                              uchar **d_inputImageGray_1, uchar **d_inputImageGray_2, uchar **d_outputImageGray,
                              float **GPU_gw_average_color_1, float **GPU_gw_auto_correlation_1,
                              float **GPU_gw_average_color_2, float **GPU_gw_auto_correlation_2,
                              float **GPU_gw_cross_correlation_3, float **GPU_gw_normalized_score_4,
                              int **match_matrix, int **disparity_map, int **depth_map,
                              //unsigned char **d_redBlurred,
                              //unsigned char **d_redBlurred_1,
                              //unsigned char **d_greenBlurred,
                              //unsigned char **d_greenBlurred_1,
                              //unsigned char **d_blueBlurred,
                              //unsigned char **d_blueBlurred_1,
                              float **h_filter, int *filterWidth,
                              const std::string &filename_1, const std::string &filename_2)
{


  //make sure the context initializes ok
  //checkCudaErrors(hipFree(0));

  cv::Mat image_1 = cv::imread(filename_1.c_str(), CV_LOAD_IMAGE_COLOR);
  cv::Mat image_2 = cv::imread(filename_2.c_str(), CV_LOAD_IMAGE_COLOR);

  if (image_1.empty() || image_2.empty())
  {
    std::cerr << "Couldn't open file: " << filename_1 << "or" << filename_2 << std::endl;
    exit(1);
  }

  //copy the transformed image from first parameter to the second one
  cv::cvtColor(image_1, G_imageInputGray_1, CV_BGR2GRAY);
  cv::cvtColor(image_2, G_imageInputGray_2, CV_BGR2GRAY);

  //allocate memory for the output
  G_imageOutputGray.create(image_1.rows, image_1.cols, CV_8UC1);

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!G_imageInputGray_1.isContinuous() || !G_imageOutputGray.isContinuous() ||
      !G_imageInputGray_2.isContinuous())
  {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *h_inputImageGray_1  = (unsigned char *)G_imageInputGray_1.ptr<unsigned char>(0);
  *h_inputImageGray_2  = (unsigned char *)G_imageInputGray_2.ptr<unsigned char>(0);
  *h_outputImageGray = (unsigned char *)G_imageOutputGray.ptr<unsigned char>(0);

  numRows = G_imageInputGray_1.rows;
  numCols = G_imageInputGray_1.cols;
  const int numPixels = numRows * numCols;

  //allocate memory on the device for both input and output
  //checkCudaErrors(   hipMalloc((void**)d_inputImageGray_1, sizeof(unsigned char) * numPixels)   );

  //hipError_t t1 = hipGetLastError();

  //cout << hipGetErrorString(t1) << endl;

  //char c;
  //cin >> c;

  checkCudaErrors(hipMalloc((void**)d_inputImageGray_1, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc((void**)d_inputImageGray_2, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc((void**)d_outputImageGray, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset((void*)*d_outputImageGray, 0,  sizeof(unsigned char) * numPixels)); //make sure no memory is left laying around

  checkCudaErrors(hipMalloc((void**)GPU_gw_average_color_1,    sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc((void**)GPU_gw_auto_correlation_1,  sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc((void**)GPU_gw_average_color_2,   sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc((void**)GPU_gw_auto_correlation_2,  sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_average_color_1,   0, sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_auto_correlation_1, 0, sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_average_color_2,  0, sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_auto_correlation_2, 0, sizeof(float) * numPixels));

  checkCudaErrors(hipMalloc((void**)GPU_gw_cross_correlation_3,   sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc((void**)GPU_gw_normalized_score_4,  sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_cross_correlation_3, 0, sizeof(float) * numPixels));
  checkCudaErrors(hipMemset((void*)*GPU_gw_normalized_score_4, 0, sizeof(float) * numPixels));

  checkCudaErrors(hipMalloc((void**)match_matrix,  sizeof(int) * numPixels));
  checkCudaErrors(hipMalloc((void**)disparity_map,  sizeof(int) * numPixels));
  checkCudaErrors(hipMalloc((void**)depth_map,  sizeof(int) * numPixels));
  checkCudaErrors(hipMemset((void*) *match_matrix, 0,  sizeof(int) * numPixels));
  checkCudaErrors(hipMemset((void*)*disparity_map, 0,  sizeof(int) * numPixels));
  checkCudaErrors(hipMemset((void*)*depth_map, 0,  sizeof(int) * numPixels));

  //copy input array to the GPU
  checkCudaErrors(hipMemcpy(*d_inputImageGray_1, *h_inputImageGray_1, sizeof(uchar) * numPixels, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(*d_inputImageGray_2, *h_inputImageGray_2, sizeof(uchar) * numPixels, hipMemcpyHostToDevice));

  d_inputImageGray_1__ = *d_inputImageGray_1;
  d_inputImageGray_2__ = *d_inputImageGray_2;
  d_outputImageGray__ = *d_outputImageGray;

  GPU_gw_average_color_1_ = *GPU_gw_average_color_1;
  GPU_gw_auto_correlation_1_ = *GPU_gw_auto_correlation_1;
  GPU_gw_average_color_2_ = *GPU_gw_average_color_2;
  GPU_gw_auto_correlation_2_ = *GPU_gw_auto_correlation_2;
  GPU_gw_cross_correlation_3_ = *GPU_gw_cross_correlation_3;
  GPU_gw_normalized_score_4_ = *GPU_gw_normalized_score_4;
  match_matrix_ = *match_matrix;
  disparity_map_ = *disparity_map;
  depth_map_ = *depth_map;



  //now create the filter that they will use
  const int blurKernelWidth = 7;
  const float blurKernelSigma = 2.;

  *filterWidth = blurKernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[blurKernelWidth * blurKernelWidth];
  h_filter__ = *h_filter;

  float filterSum = 0.f; //for normalization

  for (int r = -blurKernelWidth / 2; r <= blurKernelWidth / 2; ++r)
  {
    for (int c = -blurKernelWidth / 2; c <= blurKernelWidth / 2; ++c)
    {
      float filterValue = pow(cos(r * PI / blurKernelWidth), 2.0) * pow(cos(c * PI / blurKernelWidth), 2.0);
      //float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
      (*h_filter)[(r + blurKernelWidth / 2) * blurKernelWidth + c + blurKernelWidth / 2] = filterValue;
      //filterSum += filterValue;
    }
  }
  /*
  float normalizationFactor = 1.f / filterSum;

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
    }
  }
  //blurred
  checkCudaErrors(hipMalloc(d_redBlurred,    sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_greenBlurred,  sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_blueBlurred,   sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_redBlurred,   0, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_greenBlurred, 0, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_blueBlurred,  0, sizeof(unsigned char) * numPixels));

  */
}

/**********cleanUp**********/
void gpu_part::GPU_cleanUp(void)
{

  checkCudaErrors(hipFree(d_inputImageGray_1__));
  checkCudaErrors(hipFree(d_inputImageGray_2__));
  checkCudaErrors(hipFree(d_outputImageGray__));
  checkCudaErrors(hipFree(GPU_gw_average_color_1_));
  checkCudaErrors(hipFree(GPU_gw_auto_correlation_1_));
  checkCudaErrors(hipFree(GPU_gw_average_color_2_));
  checkCudaErrors(hipFree(GPU_gw_auto_correlation_2_));
  checkCudaErrors(hipFree(GPU_gw_cross_correlation_3_));
  checkCudaErrors(hipFree(GPU_gw_normalized_score_4_));
  checkCudaErrors(hipFree(match_matrix_));
  checkCudaErrors(hipFree(disparity_map_));
  checkCudaErrors(hipFree(depth_map_));


  delete[] h_filter__;

}

//***********************
void gpu_part::GPU_run(int argc, char **argv)
{

  uchar *h_inputImageGray_1, *h_inputImageGray_2, *d_inputImageGray_1, *d_inputImageGray_2;
  uchar *h_outputImageGray, *d_outputImageGray;
  //unsigned char *d_GrayBlurred,*d_GrayBlurred_1;//,*d_redBlurred,*d_redBlurred_1, *d_greenBlurred,*d_greenBlurred_1, *d_blueBlurred,*d_blueBlurred_1;

  float *GPU_gw_average_color_1; //  = new float[numRows * numCols];
  float *GPU_gw_auto_correlation_1; //  = new float[numRows * numCols];
  float *GPU_gw_average_color_2; //  = new float[numRows * numCols];
  float *GPU_gw_auto_correlation_2; //  = new float[numRows * numCols];
  float *GPU_gw_cross_correlation_3; //  = new float*[numRows * numCols];
  float *GPU_gw_normalized_score_4; //  = new float*[numRows * numCols];

  int *match_matrix;//=new int[numRows*numCols];
  int *disparity_map;//=new int [numRows*numCols];
  int *depth_map;//=new int [numRows*numCols];

  /*
  for (int r = 0; r < (int)numRows; ++r) {
    for (int c = 0; c < (int)numCols; ++c) {
      float image_value = static_cast<float>(channel[r * numCols + c]);
    }
  }*/

  float *h_filter;
  int    filterWidth;
  std::string input_file_1;
  std::string input_file_2;
  std::string output_file;
  std::string output_file_2;
  std::string reference_file;
  double perPixelError = 0.0;
  double globalError   = 0.0;
  bool useEpsCheck = false;
  switch (argc)
  {
  case 2:
    input_file_1 = std::string(argv[1]);
    output_file = "nwcc_output_1.png";
    output_file_2 = "nwcc_output_2.png";
    reference_file = "nwcc_reference.png";
    break;
  case 3:
    input_file_1  = std::string(argv[1]);
    input_file_2 = std::string(argv[2]);
    output_file = "nwcc_output_1.png";
    output_file_2 = "nwcc_output_2.png";
    reference_file = "nwcc_reference.png";
    break;
  case 4:
    input_file_1  = std::string(argv[1]);
    output_file = std::string(argv[2]);
    reference_file = std::string(argv[3]);
    break;
  case 6:
    useEpsCheck = true;
    input_file_1  = std::string(argv[1]);
    output_file = std::string(argv[2]);
    reference_file = std::string(argv[3]);
    perPixelError = atof(argv[4]);
    globalError   = atof(argv[5]);
    break;
  default:
    std::cerr << "Usage: ./nwcc input_file_1  input_file_2  [output_filename] [reference_filename] [perPixelError] [globalError]" << std::endl;
    exit(1);
  }

  GPU_preProcess(&h_inputImageGray_1, &h_inputImageGray_2, &h_outputImageGray,
                 &d_inputImageGray_1, &d_inputImageGray_2, &d_outputImageGray,
                 &GPU_gw_average_color_1, &GPU_gw_auto_correlation_1,
                 &GPU_gw_average_color_2, &GPU_gw_auto_correlation_2,
                 &GPU_gw_cross_correlation_3, &GPU_gw_normalized_score_4,
                 &match_matrix, &disparity_map, &depth_map,
                 &h_filter, &filterWidth, input_file_1, input_file_2);

  //load the image and give us our input and output pointers
  //preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &h_outputImageRGBA_1, &d_inputImageRGBA, &d_outputImageRGBA, &d_outputImageRGBA_1,
  //           &d_redBlurred,&d_redBlurred_1, &d_greenBlurred,&d_greenBlurred_1, &d_blueBlurred,&d_blueBlurred_1,
  //           &h_filter, &filterWidth, input_file);

  allocateMemoryAndCopyToGPU(numRows, numCols, h_filter, filterWidth);
  GpuTimer timer;
  timer.Start();

  //your_gaussian_blur(h_inputImageRGBA, d_inputImageRGBA, d_outputImageRGBA,d_outputImageRGBA_1, numRows, numCols,
  //                   d_redBlurred,d_redBlurred_1, d_greenBlurred,d_greenBlurred_1, d_blueBlurred,d_blueBlurred_1, filterWidth);

  your_gaussian_blur(h_inputImageGray_1, d_inputImageGray_1,
                     h_inputImageGray_2, d_inputImageGray_2, d_outputImageGray,
                     GPU_gw_average_color_1, GPU_gw_auto_correlation_1,
                     GPU_gw_average_color_2, GPU_gw_auto_correlation_2,
                     GPU_gw_cross_correlation_3, GPU_gw_normalized_score_4,
                     match_matrix, disparity_map, depth_map,
                     numRows, numCols, filterWidth);

  timer.Stop();
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  int err = printf("Your code ran in: %f msecs.\n", timer.Elapsed());

  if (err < 0)
  {
    //Couldn't print! Probably the student closed stdout - bad news
    std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
    exit(1);
  }

  //check results and output the blurred image
  int numPixels = numRows * numCols;
  //
  //copy the output back to the host
  checkCudaErrors(hipMemcpy(h_outputImageGray, d_outputImageGray__, sizeof(uchar) * numPixels, hipMemcpyDeviceToHost));
  //checkCudaErrors(hipMemcpy(h_outputImageRGBA_1, d_outputImageRGBA_1__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));

  GPU_postProcess(output_file, h_outputImageGray);
  //postProcess(output_file_1, h_outputImageRGBA_1);

  //GPU_referenceCalculation(h_inputImageGray_1,h_inputImageGray_2, h_outputImageGray_1,h_outputImageGray_2,numRows, numCols,h_filter, filterWidth);

  //GPU_postProcess(output_file_1, h_outputImageGray_1);
  //CPU_postProcess(output_file_2, h_outputImageGray_2);

  //CPU_postProcess(reference_file, h_outputImageRGBA);
  //  Cheater easy way with OpenCV
  //generateReferenceImage(input_file, reference_file, filterWidth);
  //compareImages(reference_file, output_file, useEpsCheck, perPixelError, globalError);

  //checkCudaErrors(hipFree(d_inputImageGray_1));
  //checkCudaErrors(hipFree(d_inputImageGray_2));
  //checkCudaErrors(hipFree(d_outputImageGray));
  //checkCudaErrors(hipFree(d_outputImageGray__));

  GPU_cleanUp();
}

/************postprocess*******/
void gpu_part::GPU_postProcess(const std::string& output_file, uchar* data_ptr)
{
  cv::Mat output(numRows, numCols, CV_8UC1, (void*)data_ptr);

  Mat img_hist_equalized;
  equalizeHist(output, img_hist_equalized);
  printf("%s\n", "hi karen5");
  //cv::Mat imageOutputBGR;
  //cv::cvtColor(output, imageOutputBGR, CV_GRAY2BGR);//CV_RGBA2BGR);
  //output the image
  //cv::imwrite(output_file.c_str(), imageOutputBGR);
  cv::imwrite(output_file.c_str(), img_hist_equalized);
}


__global__ void Kernel_1(const unsigned char* const inputChannel_1, //inputChannel
                         const unsigned char* const inputChannel_2,
                         unsigned char* const outputChannel,
                         float* const GPU_gw_average_color_1,
                         float* const GPU_gw_average_color_2,
                         float* const GPU_gw_auto_correlation_1,
                         float* const GPU_gw_auto_correlation_2,
                         int numRows, int numCols,
                         const float* const filter, const int filterWidth, int mid_shared_address)
{
  // TODO

  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int thread_1D = x + y * numCols;
  //printf("the x= %d y= %d  and  threadID %d inputvalue %d \n",x,y,thread_1D,inputChannel[thread_1D] );

  extern __shared__ unsigned char temp_1[];

  unsigned char* pointer = (unsigned char*)temp_1;
  unsigned char* temp_2 = (unsigned char*)&pointer[mid_shared_address];

  //first batch transfer
  //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
  int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
      destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
      destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
      srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
      srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), // x index in source image (-filterwidth/2 because of overlaping)
      src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image

  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );
  //int help_1 = destY*(blockDim.x+2*(filterWidth/2))+destX;
  if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
  {
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[src];
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
  }
  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (srcX < 0)
    {
      srcX = 0;
    }
    if (srcX >= numCols)
    {
      srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + srcX;

    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex];
    //printf("destY*(blockDim.x+2*(filterWidth/2))+destX= %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX);
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
  }

//Second batch loading
  dest = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
  destY = dest / (blockDim.x + 2 * (filterWidth / 2));
  destX = dest % (blockDim.x + 2 * (filterWidth / 2));
  srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2);
  srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2);
  src =  srcY * numCols + srcX;

  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );
  if (destY < (blockDim.y + 2 * (filterWidth / 2)))
  {
    //int help_2 = destY*(blockDim.x+2*(filterWidth/2))+destX;
    if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
    {
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[src];
      //printf("temp_1[help_2= %d]= %d \n",help_2,temp_1[help_2]);
    }
    else
    {
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (srcX < 0)
      {
        srcX = 0;
      }
      if (srcX >= numCols)
      {
        srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + srcX;
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex2];
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex2];
      //printf("temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX= %d]= %d , %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX,(temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]), inputChannel_2[newindex2] );

      //printf("temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX= %d]= %d , %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX,(temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX]), inputChannel_1[newindex2] );
    }
  }
  __syncthreads();

// int temp1_size=(blockDim.x+(2*(filterWidth/2)))*(blockDim.y+(2*(filterWidth/2)))*sizeof(char);

  float result_1 = 0.0f;
  float result_2 = 0.0f;
  //#pragma unroll 16
  for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
  {
    //printf("hello babak %d \n",filter_r+filterWidth/2);
    for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
    {

      float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
      float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
      float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
      result_1 += image_value_1 * filter_value;
      result_2 += image_value_2 * filter_value;
    }
  }
  result_1 = result_1 / (filterWidth * filterWidth);
  result_2 = result_2 / (filterWidth * filterWidth);
  //__syncthreads();
  if (y < numRows &&  x < numCols)
  {
    GPU_gw_average_color_1[thread_1D] = result_1;
    GPU_gw_average_color_2[thread_1D] = result_2;
  }
  __syncthreads();
  //printf("GPU_gw_average_color_1_2[thread_1D= %d]= %f , %f \n",thread_1D,GPU_gw_average_color_1[thread_1D],GPU_gw_average_color_2[thread_1D]);

// Computing the weighted auto correlation (alpha)
  result_1 = 0.0f;
  result_2 = 0.0f;
  //#pragma unroll 16

  if (y < numRows && x < numCols)
  {

    for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
    {
      //printf("hello babak %d \n",filter_r+filterWidth/2);
      for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
      {

        float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
        float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
        //float diff_Pow_2_1=(image_value_1-result_1)*(image_value_1-result_1);
        float diff_Pow_2_1 = (image_value_1 - GPU_gw_average_color_1[thread_1D]) * (image_value_1 - GPU_gw_average_color_1[thread_1D]);
        //float diff_Pow_2_2=(image_value_2-result_2)*(image_value_2-result_2);
        float diff_Pow_2_2 = (image_value_2 - GPU_gw_average_color_2[thread_1D]) * (image_value_2 - GPU_gw_average_color_2[thread_1D]);
        float filter_value_1 = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
        result_1 += filter_value_1 * diff_Pow_2_1;
        result_2 += filter_value_1 * diff_Pow_2_2;
      }
    }
    result_1 = result_1 / (filterWidth * filterWidth);
    result_2 = result_2 / (filterWidth * filterWidth);
    __syncthreads();


    GPU_gw_auto_correlation_1[thread_1D] = result_1;
    GPU_gw_auto_correlation_2[thread_1D] = result_2;
    //if (x==200){
    //printf("GPU_gw_auto_correlation_2[%d][%d]=%f\n",y,x,GPU_gw_auto_correlation_2[thread_1D]);
    //}
    //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  }
  //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  __syncthreads();
  //printf("GPU_gw_auto_correlation_1_2[thread_1D= %d]= %f , %f \n",thread_1D,GPU_gw_auto_correlation_1[thread_1D],GPU_gw_auto_correlation_2[thread_1D]);


  /*
  if ( x >= numCols || y >= numRows ){
      return;
  }
  else
  {
      outputChannel[thread_1D]=result_1;

  }*/
  //__syncthreads();
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}
//-----------------------------------
__global__ void Kernel_2(const unsigned char* const inputChannel_1, //inputChannel
                         unsigned char* const inputChannel_2,
                         unsigned char* const outputChannel,
                         float* const GPU_gw_average_color_1,
                         float* const GPU_gw_average_color_2,
                         float* const GPU_gw_auto_correlation_1,
                         float* const GPU_gw_auto_correlation_2,
                         float* const GPU_gw_cross_correlation_3,
                         float* const GPU_gw_normalized_score_4,
                         int* match_matrix,
                         int* disparity_map,
                         int* depth_map,
                         int numRows, int numCols,
                         const float* const filter, const int filterWidth, const int mid_shared_address)
{
  // TODO

  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols || absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  int x = threadIdx.x + blockIdx.x * blockDim.x; // x index of a tread in an image (not block)
  int y = threadIdx.y + blockIdx.y * blockDim.y; // y index of a tread in an image (not block)
  int thread_1D = x + y * numCols;
  //printf("the x= %d y= %d  and  threadID %d inputvalue %d \n",x,y,thread_1D,inputChannel[thread_1D] );

  extern __shared__ unsigned char temp_1[];

  unsigned char* pointer = (unsigned char*)temp_1;
  unsigned char* temp_2 = (unsigned char*)&pointer[mid_shared_address];

  //float result_1=0.0f;
  //float result_2=0.0f;
  const int max_disparity = 200;
  //int disparity; //this value used for save the number of pixels which should be compared for each pixel
  //disparity=(x>=max_disparity)?max_disparity:x;
  //disparity=((blockIdx.x*blockDim.x)>=max_disparity)?max_disparity:x;//(blockIdx.x*blockDim.x);
  if (x < numCols  && y < numRows)
  {
    GPU_gw_cross_correlation_3[thread_1D] = -100000.0; //__FLT_MIN__;
    GPU_gw_normalized_score_4[thread_1D] = -100000.0; //__FLT_MIN__;
  }
  //first batch transfer
  //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
  int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
      destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
      destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
      srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
      srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), //// x index in source image (-filterwidth/2 because of overlaping)
      src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image
  int second_srcX = srcX - max_disparity;
  int second_src = src - max_disparity; // for transfering the second window of cross correlation to shared memory

  if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
  {
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
  }
  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (srcX < 0)
    {
      srcX = 0;
    }
    if (srcX >= numCols)
    {
      srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + srcX;
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex];
  }
  // first batch loading for second window in shared memory
  //if (blockIdx.x==14 && threadIdx.y==3 ){
  //printf("second_src=%d  ",second_src);
  //}

  if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && second_srcX < numCols && second_src >= 0 && second_src < numCols * numRows)
  {
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
    //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
  }

  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (second_srcX < 0)
    {
      second_srcX = 0;
    }
    if (second_srcX >= numCols)
    {
      second_srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + second_srcX;
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
    //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
  }
//Second batch loading
  dest = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
  destY = dest / (blockDim.x + 2 * (filterWidth / 2));
  destX = dest % (blockDim.x + 2 * (filterWidth / 2));
  srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2);
  srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2);
  src =  srcY * numCols + srcX;
  second_srcX = srcX - max_disparity;
  second_src = src - max_disparity; // for transfering second batch of the second window of cross correlation to shared memory
  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,scX,src );
  if (destY < (blockDim.y + 2 * (filterWidth / 2)))
  {
    //int help_2 = destY*(blockDim.x+2*(filterWidth/2))+destX;
    if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
    {
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
    }
    else
    {
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (srcX < 0)
      {
        srcX = 0;
      }
      if (srcX >= numCols)
      {
        srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + srcX;
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex2];
    }
    // second batch loading for second window in shared memory

    if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && second_srcX < numCols  && second_src >= 0 && second_src < numCols * numRows)
    {
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
      //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
    }
    else
    {
      //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
      //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (second_srcX < 0)
      {
        second_srcX = 0;
      }
      if (second_srcX >= numCols)
      {
        second_srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + second_srcX;
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex2];
      //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
    }
  }
  __syncthreads();


  //until here there was just transfering the first block from main memory to shared memoryhelecoptrei zadi,
  //after here it is computing the convultion filter and updating the shared memory

  for (int j = 0; j < max_disparity; j++)
  {

    if (x - max_disparity + j >= 0)
    {
      float result_1 = 0.0f;
      if (y >= 0 && y < numRows && x >= 0 && x < numCols && (thread_1D - max_disparity + j) >= 0)
      {
        for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
        {
          //printf("hello babak %d \n",filter_r+filterWidth/2);
          for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
          {
            float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
            float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
            //if (x==100 && y==100 && j==50){
            //printf(" hello babak2 x=%d y=%d j=%d  image_value_1=%f image_value_2=%f  \n", x,y,j,image_value_1,image_value_2 );
            //}
            float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
            float image_minus_average_1 = static_cast<float>(image_value_1) - static_cast<float>(GPU_gw_average_color_1[thread_1D]);
            float image_minus_average_2 = static_cast<float>(image_value_2) - static_cast<float>(GPU_gw_average_color_2[thread_1D - max_disparity + j]);
            result_1 += image_minus_average_1 * image_minus_average_2 * filter_value;
          }
        }
        //__syncthreads();
        result_1 = result_1 / (filterWidth * filterWidth);
        //printf("result_1,GPU_gw_cross_correlation_3[thread_1D=%d]=%f   %f\n",thread_1D,result_1,GPU_gw_cross_correlation_3[thread_1D]);
        float help_3 = (static_cast<float>(result_1) / (static_cast<float>(sqrt(GPU_gw_auto_correlation_1[thread_1D] * GPU_gw_auto_correlation_2[thread_1D - max_disparity + j]))));
        //__syncthreads();
        if (help_3 > GPU_gw_normalized_score_4[thread_1D] && y < numRows && x < numCols) // && (x-max_disparity+j)>=0 ){
        {
          GPU_gw_cross_correlation_3[thread_1D] = result_1;
          GPU_gw_normalized_score_4[thread_1D] = help_3;
          match_matrix[thread_1D] = x - max_disparity + j;
          //disparity_map[thread_1D]=abs(match_matrix[thread_1D]-x);
          //match_matrix[thread_1D]=x-(x-max_disparity+j);
          //if (x==100 && y==100){
          //printf("x=%d y=%d j=%d thread_1D=%d help_3=%f  match_matrix[thread_1D]=%d  \n", x,y,j,thread_1D,help_3,match_matrix[thread_1D]);
          //}
        }
      }
    }
    // moving second matching window
    //first batch transfer
    //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
    unsigned char help_1, help_2;
    int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
        destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
        destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
        srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
        srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), //// x index in source image (-filterwidth/2 because of overlaping)
        src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image
    int second_srcX = srcX - max_disparity + (j + 1);
    int second_src = src - max_disparity + (j + 1); // for transfering the second window of cross correlation to shared memory
    //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
    //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );

    // first batch moving for second window in shared memory

    if (destX >= 1 && destX < (blockDim.x + 2 * (filterWidth / 2)) && destY < (blockDim.y + 2 * (filterWidth / 2)))
    {
      //if(blockIdx.x==4 && blockIdx.y==0 && j==190 ){
      //printf(" hello booob2 dest=%d x=%d y=%d threadIdx.x=%d threadIdx.y=%d j=%d destX=%d destY=%d \n", dest,x,y,threadIdx.x,threadIdx.y,j,destX,destY);
      //}
      help_1 = temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX];
    }
    syncthreads();
    if (destX >= 1 && destX < (blockDim.x + 2 * (filterWidth / 2)) && destY < (blockDim.y + 2 * (filterWidth / 2)))
    {
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX - 1] = help_1;
    }
    //__syncthreads();
    if ((destX == ((blockDim.x + 2 * (filterWidth / 2)) - 1)) && destY < (blockDim.y + 2 * (filterWidth / 2))  && second_srcX >= 0)
    {

      //if (blockIdx.x==5 && blockIdx.y==5){
      //    printf(" hello booob x=%d y=%d j=%d destX=%d destY=%d \n", x,y,j,destX,destY);
      //}
      if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && (second_srcX + 1) < numCols  && second_src >= 0 && second_src < numCols * numRows)
      {
        temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
      }
      else
      {
        //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
        //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
        if (srcY < 0)
        {
          srcY = 0;
        }
        if (srcY >= numRows)
        {
          srcY = numRows - 1 ;
        }
        if (second_srcX < 0)
        {
          second_srcX = 0;
        }
        if ((second_srcX) >= numCols)
        {
          second_srcX = numCols - 1 ;
        }
        int newindex = srcY * numCols + second_srcX;
        temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
      }
    }
    __syncthreads();

    //Second batch moving
    int dest_2 = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
    int destY_2 = dest_2 / (blockDim.x + 2 * (filterWidth / 2));
    int destX_2 = dest_2 % (blockDim.x + 2 * (filterWidth / 2));
    int srcY_2 = blockIdx.y * blockDim.y + destY_2 - (filterWidth / 2);
    int srcX_2 = blockIdx.x * blockDim.x + destX_2 - (filterWidth / 2);
    int src_2 =  srcY_2 * numCols + srcX_2;
    int second_srcX_2 = srcX_2 - max_disparity + (j + 1);;
    int second_src_2 = src_2 - max_disparity + (j + 1); // for transfering second batch of the second window of cross correlation to shared memory
    //if(blockIdx.x==0 && blockIdx.y==0   && destY_2 < (blockDim.y+2*(filterWidth/2))  ){
    //    printf(" hello booob3 dest_2=%d x=%d y=%d threadIdx.x=%d threadIdx.y=%d j=%d destX_2=%d destY_2=%d \n", dest_2,x,y,threadIdx.x,threadIdx.y,j,destX_2,destY_2);
    //}
    //if(destY_2 < (blockDim.y+2*(filterWidth/2))){

    ///*

    if (destX_2 >= 1 && destX_2 < (blockDim.x + 2 * (filterWidth / 2)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)))
    {
      help_2 = temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2];
    }

    syncthreads();
    if (destX_2 >= 1 && destX_2 < (blockDim.x + 2 * (filterWidth / 2)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)))
    {
      temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2 - 1] = help_2;
    }
    //__syncthreads();
    //printf("destX=%d ((blockDim.x+2*(filterWidth/2))-1)=%d destY=%d (blockDim.y+2*(filterWidth/2))=%d \n",destX ,((blockDim.x+2*(filterWidth/2))-1),destY,(blockDim.y+2*(filterWidth/2)));

    //if ((destX== ((blockDim.x+2*(filterWidth/2))-1))){
    //  printf("destX=%d destY=%d \n", destX,destY);
    //}
    if ((destX_2 == ((blockDim.x + 2 * (filterWidth / 2)) - 1)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)) && second_srcX_2 >= 0)
    {
      //if (x==100 && y==100){
      //  printf(" hello booob x=%d y=%d j=%d \n", x,y,j);
      //}
      //printf("destX_2=%d destY_2=%d \n", destX_2,destY_2);
      if (srcY_2 >= 0 && srcY_2 < numRows && second_srcX_2 >= 0 && (second_srcX_2 + 1) < numCols  && second_src_2 >= 0 && second_src_2 < numCols * numRows)
      {
        temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2] = inputChannel_2[second_src_2];
      }
      else
      {
        //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
        //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
        if (srcY_2 < 0)
        {
          srcY_2 = 0;
        }
        if (srcY_2 >= numRows)
        {
          srcY_2 = numRows - 1 ;
        }
        if (second_srcX_2 < 0)
        {
          second_srcX_2 = 0;
        }
        if ((second_srcX_2) >= numCols)
        {
          second_srcX_2 = numCols - 1 ;
        }
        int newindex2_2 = srcY_2 * numCols + second_srcX_2;
        temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2] = inputChannel_2[newindex2_2];
      }
    }//*/
    //__syncthreads();
/////////////////////////////////////////////////////////////////
  }

  __syncthreads();
  if (y < numRows && y > 0 && x > 0 && x < numCols)
  {
    disparity_map[thread_1D] = abs(match_matrix[thread_1D] - x);
    //printf("disparity_map[%d]= %d\n",thread_1D,disparity_map[thread_1D]);
    //__syncthreads();
    //outputChannel[thread_1D] =static_cast<unsigned char>(GPU_gw_average_color_2[thread_1D]);
    outputChannel[thread_1D] = static_cast<unsigned char>(disparity_map[thread_1D]);
  }
  //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  __syncthreads();

  //---------------------------------------------------------------------------------------------------------

  /*
  if ( x >= numCols || y >= numRows ){
      return;
  }
  else
  {
      outputChannel[thread_1D]=result_1;

  }*/

  //__syncthreads();
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}



float         *d_filter;

void gpu_part::allocateMemoryAndCopyToGPU(const int numRows, const int numCols,
    const float* const h_filter, const int filterWidth)
{

  //allocate memory for the three different channels
  //original


  //checkCudaErrors(hipMalloc(&d_Gray_1,   sizeof(unsigned char) * numRows * numCols));
  //checkCudaErrors(hipMalloc(&d_Gray_2,   sizeof(unsigned char) * numRows * numCols));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float)*filterWidth * filterWidth));


  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  hipMemcpy(d_filter, h_filter, sizeof(float)*filterWidth * filterWidth, hipMemcpyHostToDevice);

}



void gpu_part::your_gaussian_blur(const unsigned char* const h_inputImageGray_1,
                                  unsigned char* const d_inputImageGray_1,
                                  unsigned char* const h_inputImageGray_2,
                                  unsigned char* const d_inputImageGray_2,
                                  unsigned char* const d_outputImageGray,
                                  float* const GPU_gw_average_color_1,
                                  float* const GPU_gw_auto_correlation_1,
                                  float* const GPU_gw_average_color_2,
                                  float* const GPU_gw_auto_correlation_2,
                                  float* const GPU_gw_cross_correlation_3,
                                  float* const GPU_gw_normalized_score_4,
                                  int* match_matrix,
                                  int* disparity_map,
                                  int* depth_map,
                                  const int numRows, const int numCols,
                                  const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(32, 32);
  //const dim3 blockSize(8,8);
  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  //const dim3 gridSize(15,12);
  //const dim3 gridSize(5,1);
  //const dim3 gridSize(1,1);
  const dim3 gridSize(ceilf(static_cast<float>(numCols) / blockSize.x), ceilf(static_cast<float>(numRows) / blockSize.y));

  //TODO: Launch a kernel for separating the RGBA image into different color channels
  //separateChannels<<<gridSize,blockSize>>>(d_inputImageRGBA,(int)numRows,(int)numCols,d_red, d_green,d_blue);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  //hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //TODO: Call your convolution kernel here 3 times, once for each color channel.
  int mid_shared_address = (blockSize.x + (2 * (filterWidth / 2))) * (blockSize.y + (2 * (filterWidth / 2))) * sizeof(unsigned char);
  Kernel_1 <<< gridSize, blockSize, (blockSize.x + (2 * (filterWidth / 2)))*(blockSize.y + (2 * (filterWidth / 2)))*sizeof(unsigned char) * 2 >>> (d_inputImageGray_1,
      d_inputImageGray_2,
      d_outputImageGray,
      GPU_gw_average_color_1,
      GPU_gw_average_color_2,
      GPU_gw_auto_correlation_1,
      GPU_gw_auto_correlation_2,
      (int)numRows,
      (int)numCols,
      d_filter,
      filterWidth, mid_shared_address);
  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  //for (int r = 0; r < (int)numRows; ++r) {
  //    for (int c = 0; c < (int)numCols; ++c) {
  //     printf("%f ",GPU_gw_auto_correlation_1[r * numCols + c]);
  //    }
  //    printf("\n");
  //}


  Kernel_2 <<< gridSize, blockSize, (blockSize.x + (2 * (filterWidth / 2)))*(blockSize.y + (2 * (filterWidth / 2)))*sizeof(char) * 2 >>> (d_inputImageGray_1,
      d_inputImageGray_2,
      d_outputImageGray,
      GPU_gw_average_color_1,
      GPU_gw_average_color_2,
      GPU_gw_auto_correlation_1,
      GPU_gw_auto_correlation_2,
      GPU_gw_cross_correlation_3,
      GPU_gw_normalized_score_4,
      match_matrix,
      disparity_map,
      depth_map,
      (int)numRows,
      (int)numCols,
      d_filter,
      filterWidth, mid_shared_address);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  /*


   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

   // Now we recombine your results. We take care of launching this kernel for you.
   //
   // NOTE: This kernel launch depends on the gridSize and blockSize variables,
   // which you must set yourself.
   recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                              d_greenBlurred,
                                              d_blueBlurred,
                                              d_outputImageRGBA,
                                              numRows,
                                              numCols);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


   recombineChannels<<<gridSize, blockSize>>>(d_redBlurred_1,
                                              d_greenBlurred_1,
                                              d_blueBlurred_1,
                                              d_outputImageRGBA_1,
                                              numRows,
                                              numCols);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  */

}


}
}
}
}
}
