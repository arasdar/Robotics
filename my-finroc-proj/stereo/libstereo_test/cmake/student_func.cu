#include "hip/hip_runtime.h"
// code written by babak ebrahimi summer 2014
//#include "reference_calc.cpp"
#include <algorithm>
#include <cassert>
// for uchar4 struct
#include <hip/hip_runtime.h>
#include "utils.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

__global__ void Kernel_1(const unsigned char* const inputChannel_1, //inputChannel
                         const unsigned char* const inputChannel_2,
                         unsigned char* const outputChannel,
                         float* const GPU_gw_average_color_1,
                         float* const GPU_gw_average_color_2,
                         float* const GPU_gw_auto_correlation_1,
                         float* const GPU_gw_auto_correlation_2,
                         int numRows, int numCols,
                         const float* const filter, const int filterWidth, int mid_shared_address)
{
  // TODO

  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int thread_1D = x + y * numCols;
  //printf("the x= %d y= %d  and  threadID %d inputvalue %d \n",x,y,thread_1D,inputChannel[thread_1D] );

  extern __shared__ unsigned char temp_1[];

  unsigned char* pointer = (unsigned char*)temp_1;
  unsigned char* temp_2 = (unsigned char*)&pointer[mid_shared_address];

  //first batch transfer
  //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
  int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
      destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
      destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
      srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
      srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), // x index in source image (-filterwidth/2 because of overlaping)
      src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image

  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );
  //int help_1 = destY*(blockDim.x+2*(filterWidth/2))+destX;
  if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
  {
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[src];
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
  }
  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (srcX < 0)
    {
      srcX = 0;
    }
    if (srcX >= numCols)
    {
      srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + srcX;

    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex];
    //printf("destY*(blockDim.x+2*(filterWidth/2))+destX= %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX);
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
  }

//Second batch loading
  dest = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
  destY = dest / (blockDim.x + 2 * (filterWidth / 2));
  destX = dest % (blockDim.x + 2 * (filterWidth / 2));
  srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2);
  srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2);
  src =  srcY * numCols + srcX;

  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );
  if (destY < (blockDim.y + 2 * (filterWidth / 2)))
  {
    //int help_2 = destY*(blockDim.x+2*(filterWidth/2))+destX;
    if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
    {
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[src];
      //printf("temp_1[help_2= %d]= %d \n",help_2,temp_1[help_2]);
    }
    else
    {
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (srcX < 0)
      {
        srcX = 0;
      }
      if (srcX >= numCols)
      {
        srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + srcX;
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex2];
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex2];
      //printf("temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX= %d]= %d , %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX,(temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]), inputChannel_2[newindex2] );

      //printf("temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX= %d]= %d , %d \n",destY*(blockDim.x+2*(filterWidth/2))+destX,(temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX]), inputChannel_1[newindex2] );
    }
  }
  __syncthreads();

// int temp1_size=(blockDim.x+(2*(filterWidth/2)))*(blockDim.y+(2*(filterWidth/2)))*sizeof(char);

  float result_1 = 0.0f;
  float result_2 = 0.0f;
  //#pragma unroll 16
  for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
  {
    //printf("hello babak %d \n",filter_r+filterWidth/2);
    for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
    {

      float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
      float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
      float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
      result_1 += image_value_1 * filter_value;
      result_2 += image_value_2 * filter_value;
    }
  }
  result_1 = result_1 / (filterWidth * filterWidth);
  result_2 = result_2 / (filterWidth * filterWidth);
  //__syncthreads();
  if (y < numRows &&  x < numCols)
  {
    GPU_gw_average_color_1[thread_1D] = result_1;
    GPU_gw_average_color_2[thread_1D] = result_2;
  }
  __syncthreads();
  //printf("GPU_gw_average_color_1_2[thread_1D= %d]= %f , %f \n",thread_1D,GPU_gw_average_color_1[thread_1D],GPU_gw_average_color_2[thread_1D]);

// Computing the weighted auto correlation (alpha)
  result_1 = 0.0f;
  result_2 = 0.0f;
  //#pragma unroll 16

  if (y < numRows && x < numCols)
  {

    for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
    {
      //printf("hello babak %d \n",filter_r+filterWidth/2);
      for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
      {

        float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
        float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
        //float diff_Pow_2_1=(image_value_1-result_1)*(image_value_1-result_1);
        float diff_Pow_2_1 = (image_value_1 - GPU_gw_average_color_1[thread_1D]) * (image_value_1 - GPU_gw_average_color_1[thread_1D]);
        //float diff_Pow_2_2=(image_value_2-result_2)*(image_value_2-result_2);
        float diff_Pow_2_2 = (image_value_2 - GPU_gw_average_color_2[thread_1D]) * (image_value_2 - GPU_gw_average_color_2[thread_1D]);
        float filter_value_1 = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
        result_1 += filter_value_1 * diff_Pow_2_1;
        result_2 += filter_value_1 * diff_Pow_2_2;
      }
    }
    result_1 = result_1 / (filterWidth * filterWidth);
    result_2 = result_2 / (filterWidth * filterWidth);
    __syncthreads();


    GPU_gw_auto_correlation_1[thread_1D] = result_1;
    GPU_gw_auto_correlation_2[thread_1D] = result_2;
    //if (x==200){
    //printf("GPU_gw_auto_correlation_2[%d][%d]=%f\n",y,x,GPU_gw_auto_correlation_2[thread_1D]);
    //}
    //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  }
  //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  __syncthreads();
  //printf("GPU_gw_auto_correlation_1_2[thread_1D= %d]= %f , %f \n",thread_1D,GPU_gw_auto_correlation_1[thread_1D],GPU_gw_auto_correlation_2[thread_1D]);


  /*
  if ( x >= numCols || y >= numRows ){
      return;
  }
  else
  {
      outputChannel[thread_1D]=result_1;

  }*/
  //__syncthreads();
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}
//-----------------------------------
__global__ void Kernel_2(const unsigned char* const inputChannel_1, //inputChannel
                         unsigned char* const inputChannel_2,
                         unsigned char* const outputChannel,
                         float* const GPU_gw_average_color_1,
                         float* const GPU_gw_average_color_2,
                         float* const GPU_gw_auto_correlation_1,
                         float* const GPU_gw_auto_correlation_2,
                         float* const GPU_gw_cross_correlation_3,
                         float* const GPU_gw_normalized_score_4,
                         int* match_matrix,
                         int* disparity_map,
                         int* depth_map,
                         int numRows, int numCols,
                         const float* const filter, const int filterWidth, const int mid_shared_address)
{
  // TODO

  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols || absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
  int x = threadIdx.x + blockIdx.x * blockDim.x; // x index of a tread in an image (not block)
  int y = threadIdx.y + blockIdx.y * blockDim.y; // y index of a tread in an image (not block)
  int thread_1D = x + y * numCols;
  //printf("the x= %d y= %d  and  threadID %d inputvalue %d \n",x,y,thread_1D,inputChannel[thread_1D] );

  extern __shared__ unsigned char temp_1[];

  unsigned char* pointer = (unsigned char*)temp_1;
  unsigned char* temp_2 = (unsigned char*)&pointer[mid_shared_address];

  //float result_1=0.0f;
  //float result_2=0.0f;
  const int max_disparity = 200;
  //int disparity; //this value used for save the number of pixels which should be compared for each pixel
  //disparity=(x>=max_disparity)?max_disparity:x;
  //disparity=((blockIdx.x*blockDim.x)>=max_disparity)?max_disparity:x;//(blockIdx.x*blockDim.x);
  if (x < numCols  && y < numRows)
  {
    GPU_gw_cross_correlation_3[thread_1D] = -100000.0; //__FLT_MIN__;
    GPU_gw_normalized_score_4[thread_1D] = -100000.0; //__FLT_MIN__;
  }
  //first batch transfer
  //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
  int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
      destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
      destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
      srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
      srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), //// x index in source image (-filterwidth/2 because of overlaping)
      src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image
  int second_srcX = srcX - max_disparity;
  int second_src = src - max_disparity; // for transfering the second window of cross correlation to shared memory

  if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
  {
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
  }
  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (srcX < 0)
    {
      srcX = 0;
    }
    if (srcX >= numCols)
    {
      srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + srcX;
    temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex];
  }
  // first batch loading for second window in shared memory
  //if (blockIdx.x==14 && threadIdx.y==3 ){
  //printf("second_src=%d  ",second_src);
  //}

  if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && second_srcX < numCols && second_src >= 0 && second_src < numCols * numRows)
  {
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
    //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
  }

  else
  {
    //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
    //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
    if (srcY < 0)
    {
      srcY = 0;
    }
    if (srcY >= numRows)
    {
      srcY = numRows - 1 ;
    }
    if (second_srcX < 0)
    {
      second_srcX = 0;
    }
    if (second_srcX >= numCols)
    {
      second_srcX = numCols - 1 ;
    }
    int newindex = srcY * numCols + second_srcX;
    temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
    //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
  }
//Second batch loading
  dest = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
  destY = dest / (blockDim.x + 2 * (filterWidth / 2));
  destX = dest % (blockDim.x + 2 * (filterWidth / 2));
  srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2);
  srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2);
  src =  srcY * numCols + srcX;
  second_srcX = srcX - max_disparity;
  second_src = src - max_disparity; // for transfering second batch of the second window of cross correlation to shared memory
  //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
  //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,scX,src );
  if (destY < (blockDim.y + 2 * (filterWidth / 2)))
  {
    //int help_2 = destY*(blockDim.x+2*(filterWidth/2))+destX;
    if (srcY >= 0 && srcY < numRows && srcX >= 0 && srcX < numCols)
    {
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[src];
    }
    else
    {
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (srcX < 0)
      {
        srcX = 0;
      }
      if (srcX >= numCols)
      {
        srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + srcX;
      temp_1[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_1[newindex2];
    }
    // second batch loading for second window in shared memory

    if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && second_srcX < numCols  && second_src >= 0 && second_src < numCols * numRows)
    {
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
      //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
    }
    else
    {
      //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
      //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
      if (srcY < 0)
      {
        srcY = 0;
      }
      if (srcY >= numRows)
      {
        srcY = numRows - 1 ;
      }
      if (second_srcX < 0)
      {
        second_srcX = 0;
      }
      if (second_srcX >= numCols)
      {
        second_srcX = numCols - 1 ;
      }
      int newindex2 = srcY * numCols + second_srcX;
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex2];
      //printf("temp2[%d]=%d\t",destY*(blockDim.x+2*(filterWidth/2))+destX,temp_2[destY*(blockDim.x+2*(filterWidth/2))+destX]);
    }
  }
  __syncthreads();


  //until here there was just transfering the first block from main memory to shared memoryhelecoptrei zadi,
  //after here it is computing the convultion filter and updating the shared memory

  for (int j = 0; j < max_disparity; j++)
  {

    if (x - max_disparity + j >= 0)
    {
      float result_1 = 0.0f;
      if (y >= 0 && y < numRows && x >= 0 && x < numCols && (thread_1D - max_disparity + j) >= 0)
      {
        for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; filter_r++)
        {
          //printf("hello babak %d \n",filter_r+filterWidth/2);
          for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; filter_c++)
          {
            float image_value_1 = static_cast<float>(temp_1[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
            float image_value_2 = static_cast<float>(temp_2[(threadIdx.y + filter_r + filterWidth / 2) * (blockDim.x + 2 * (filterWidth / 2)) + threadIdx.x + filter_c + filterWidth / 2]);
            //if (x==100 && y==100 && j==50){
            //printf(" hello babak2 x=%d y=%d j=%d  image_value_1=%f image_value_2=%f  \n", x,y,j,image_value_1,image_value_2 );
            //}
            float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];
            float image_minus_average_1 = static_cast<float>(image_value_1) - static_cast<float>(GPU_gw_average_color_1[thread_1D]);
            float image_minus_average_2 = static_cast<float>(image_value_2) - static_cast<float>(GPU_gw_average_color_2[thread_1D - max_disparity + j]);
            result_1 += image_minus_average_1 * image_minus_average_2 * filter_value;
          }
        }
        //__syncthreads();
        result_1 = result_1 / (filterWidth * filterWidth);
        //printf("result_1,GPU_gw_cross_correlation_3[thread_1D=%d]=%f   %f\n",thread_1D,result_1,GPU_gw_cross_correlation_3[thread_1D]);
        float help_3 = (static_cast<float>(result_1) / (static_cast<float>(sqrt(GPU_gw_auto_correlation_1[thread_1D] * GPU_gw_auto_correlation_2[thread_1D - max_disparity + j]))));
        //__syncthreads();
        if (help_3 > GPU_gw_normalized_score_4[thread_1D] && y < numRows && x < numCols) // && (x-max_disparity+j)>=0 ){
        {
          GPU_gw_cross_correlation_3[thread_1D] = result_1;
          GPU_gw_normalized_score_4[thread_1D] = help_3;
          match_matrix[thread_1D] = x - max_disparity + j;
          //disparity_map[thread_1D]=abs(match_matrix[thread_1D]-x);
          //match_matrix[thread_1D]=x-(x-max_disparity+j);
          //if (x==100 && y==100){
          //printf("x=%d y=%d j=%d thread_1D=%d help_3=%f  match_matrix[thread_1D]=%d  \n", x,y,j,thread_1D,help_3,match_matrix[thread_1D]);
          //}
        }
      }
    }
    // moving second matching window
    //first batch transfer
    //temp_1[(threadIdx.x+filterWidth/2)+(threadIdx.y+filterWidth/2)*(blockDim.x+2*(filterWidth/2))]=inputChannel[thread_1D];
    unsigned char help_1, help_2;
    int dest = threadIdx.y * blockDim.x + threadIdx.x, // index(in a one dimentional array) of a thread(pixel) in a block
        destY = dest / (blockDim.x + 2 * (filterWidth / 2)), // y index of destination in shared memory
        destX = dest % (blockDim.x + 2 * (filterWidth / 2)), // x index of destination in shared memory
        srcY = blockIdx.y * blockDim.y + destY - (filterWidth / 2), // y index in source image (-filterwidth/2 because of overlaping)
        srcX = blockIdx.x * blockDim.x + destX - (filterWidth / 2), //// x index in source image (-filterwidth/2 because of overlaping)
        src = srcY * numCols + srcX;// index(in a one dimentional array) of a thread(pixel) in the source image
    int second_srcX = srcX - max_disparity + (j + 1);
    int second_src = src - max_disparity + (j + 1); // for transfering the second window of cross correlation to shared memory
    //printf("threadIdx.x= %d threadIdx.y= %d blockIdx.x= %d blockIdx.y= %d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y);
    //printf("dest= %d destY= %d destX= %d srcY= %d srcX= %d src %d \n",dest,destY,destX,srcY,srcX,src );

    // first batch moving for second window in shared memory

    if (destX >= 1 && destX < (blockDim.x + 2 * (filterWidth / 2)) && destY < (blockDim.y + 2 * (filterWidth / 2)))
    {
      //if(blockIdx.x==4 && blockIdx.y==0 && j==190 ){
      //printf(" hello booob2 dest=%d x=%d y=%d threadIdx.x=%d threadIdx.y=%d j=%d destX=%d destY=%d \n", dest,x,y,threadIdx.x,threadIdx.y,j,destX,destY);
      //}
      help_1 = temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX];
    }
    syncthreads();
    if (destX >= 1 && destX < (blockDim.x + 2 * (filterWidth / 2)) && destY < (blockDim.y + 2 * (filterWidth / 2)))
    {
      temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX - 1] = help_1;
    }
    //__syncthreads();
    if ((destX == ((blockDim.x + 2 * (filterWidth / 2)) - 1)) && destY < (blockDim.y + 2 * (filterWidth / 2))  && second_srcX >= 0)
    {

      //if (blockIdx.x==5 && blockIdx.y==5){
      //    printf(" hello booob x=%d y=%d j=%d destX=%d destY=%d \n", x,y,j,destX,destY);
      //}
      if (srcY >= 0 && srcY < numRows && second_srcX >= 0 && (second_srcX + 1) < numCols  && second_src >= 0 && second_src < numCols * numRows)
      {
        temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[second_src];
      }
      else
      {
        //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
        //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
        if (srcY < 0)
        {
          srcY = 0;
        }
        if (srcY >= numRows)
        {
          srcY = numRows - 1 ;
        }
        if (second_srcX < 0)
        {
          second_srcX = 0;
        }
        if ((second_srcX) >= numCols)
        {
          second_srcX = numCols - 1 ;
        }
        int newindex = srcY * numCols + second_srcX;
        temp_2[destY * (blockDim.x + 2 * (filterWidth / 2)) + destX] = inputChannel_2[newindex];
      }
    }
    __syncthreads();

    //Second batch moving
    int dest_2 = threadIdx.y * blockDim.x + threadIdx.x + blockDim.x * blockDim.y;
    int destY_2 = dest_2 / (blockDim.x + 2 * (filterWidth / 2));
    int destX_2 = dest_2 % (blockDim.x + 2 * (filterWidth / 2));
    int srcY_2 = blockIdx.y * blockDim.y + destY_2 - (filterWidth / 2);
    int srcX_2 = blockIdx.x * blockDim.x + destX_2 - (filterWidth / 2);
    int src_2 =  srcY_2 * numCols + srcX_2;
    int second_srcX_2 = srcX_2 - max_disparity + (j + 1);;
    int second_src_2 = src_2 - max_disparity + (j + 1); // for transfering second batch of the second window of cross correlation to shared memory
    //if(blockIdx.x==0 && blockIdx.y==0   && destY_2 < (blockDim.y+2*(filterWidth/2))  ){
    //    printf(" hello booob3 dest_2=%d x=%d y=%d threadIdx.x=%d threadIdx.y=%d j=%d destX_2=%d destY_2=%d \n", dest_2,x,y,threadIdx.x,threadIdx.y,j,destX_2,destY_2);
    //}
    //if(destY_2 < (blockDim.y+2*(filterWidth/2))){

    ///*

    if (destX_2 >= 1 && destX_2 < (blockDim.x + 2 * (filterWidth / 2)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)))
    {
      help_2 = temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2];
    }

    syncthreads();
    if (destX_2 >= 1 && destX_2 < (blockDim.x + 2 * (filterWidth / 2)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)))
    {
      temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2 - 1] = help_2;
    }
    //__syncthreads();
    //printf("destX=%d ((blockDim.x+2*(filterWidth/2))-1)=%d destY=%d (blockDim.y+2*(filterWidth/2))=%d \n",destX ,((blockDim.x+2*(filterWidth/2))-1),destY,(blockDim.y+2*(filterWidth/2)));

    //if ((destX== ((blockDim.x+2*(filterWidth/2))-1))){
    //  printf("destX=%d destY=%d \n", destX,destY);
    //}
    if ((destX_2 == ((blockDim.x + 2 * (filterWidth / 2)) - 1)) && destY_2 < (blockDim.y + 2 * (filterWidth / 2)) && second_srcX_2 >= 0)
    {
      //if (x==100 && y==100){
      //  printf(" hello booob x=%d y=%d j=%d \n", x,y,j);
      //}
      //printf("destX_2=%d destY_2=%d \n", destX_2,destY_2);
      if (srcY_2 >= 0 && srcY_2 < numRows && second_srcX_2 >= 0 && (second_srcX_2 + 1) < numCols  && second_src_2 >= 0 && second_src_2 < numCols * numRows)
      {
        temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2] = inputChannel_2[second_src_2];
      }
      else
      {
        //temp_1[destY*(blockDim.x+2*(filterWidth/2))+destX] =0; //;
        //printf("temp_1[help_1=%d]=%d \n",help_1,temp_1[help_1]);
        if (srcY_2 < 0)
        {
          srcY_2 = 0;
        }
        if (srcY_2 >= numRows)
        {
          srcY_2 = numRows - 1 ;
        }
        if (second_srcX_2 < 0)
        {
          second_srcX_2 = 0;
        }
        if ((second_srcX_2) >= numCols)
        {
          second_srcX_2 = numCols - 1 ;
        }
        int newindex2_2 = srcY_2 * numCols + second_srcX_2;
        temp_2[destY_2 * (blockDim.x + 2 * (filterWidth / 2)) + destX_2] = inputChannel_2[newindex2_2];
      }
    }//*/
    //__syncthreads();
/////////////////////////////////////////////////////////////////
  }

  __syncthreads();
  if (y < numRows && y > 0 && x > 0 && x < numCols)
  {
    disparity_map[thread_1D] = abs(match_matrix[thread_1D] - x);
    //printf("disparity_map[%d]= %d\n",thread_1D,disparity_map[thread_1D]);
    //__syncthreads();
    //outputChannel[thread_1D] =static_cast<unsigned char>(GPU_gw_average_color_2[thread_1D]);
    outputChannel[thread_1D] = static_cast<unsigned char>(disparity_map[thread_1D]);
  }
  //outputChannel[thread_1D]=GPU_gw_auto_correlation_1[thread_1D];
  __syncthreads();

  //---------------------------------------------------------------------------------------------------------

  /*
  if ( x >= numCols || y >= numRows ){
      return;
  }
  else
  {
      outputChannel[thread_1D]=result_1;

  }*/

  //__syncthreads();
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}



//------------------------------

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__ void separateChannels(const uchar4* const inputImageRGBA,
                                 int numRows,
                                 int numCols,
                                 unsigned char* const redChannel,
                                 unsigned char* const greenChannel,
                                 unsigned char* const blueChannel)
{
  // TODO
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  int absolute_image_position_x = threadIdx.x + blockIdx.x * blockDim.x;
  int absolute_image_position_y = threadIdx.y + blockIdx.y * blockDim.y;
  if (absolute_image_position_x >= numCols ||
      absolute_image_position_y >= numRows)
  {
    return;
  }
  int i = numCols * absolute_image_position_y + absolute_image_position_x;
  redChannel[i] = inputImageRGBA[i].x;
  greenChannel[i] = inputImageRGBA[i].y;
  blueChannel[i] = inputImageRGBA[i].z;
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__ void recombineChannels(const unsigned char* const redChannel,
                                  const unsigned char* const greenChannel,
                                  const unsigned char* const blueChannel,
                                  uchar4* const outputImageRGBA,
                                  int numRows,
                                  int numCols)
{
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}
/**
unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter,sizeof(float)*(int)filterWidth*(int)filterWidth));


  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  hipMemcpy(d_filter,h_filter,sizeof(float)*(int)filterWidth*(int)filterWidth,hipMemcpyHostToDevice);

}*/

//unsigned char *d_red,*d_green,*d_blue;
//unsigned char *d_Gray_1,*d_Gray_2;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const int numRows, const int numCols,
                                const float* const h_filter, const int filterWidth)
{

  //allocate memory for the three different channels
  //original


  //checkCudaErrors(hipMalloc(&d_Gray_1,   sizeof(unsigned char) * numRows * numCols));
  //checkCudaErrors(hipMalloc(&d_Gray_2,   sizeof(unsigned char) * numRows * numCols));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float)*filterWidth * filterWidth));


  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  hipMemcpy(d_filter, h_filter, sizeof(float)*filterWidth * filterWidth, hipMemcpyHostToDevice);

}



void your_gaussian_blur(const unsigned char* const h_inputImageGray_1,
                        unsigned char* const d_inputImageGray_1,
                        unsigned char* const h_inputImageGray_2,
                        unsigned char* const d_inputImageGray_2,
                        unsigned char* const d_outputImageGray,
                        float* const GPU_gw_average_color_1,
                        float* const GPU_gw_auto_correlation_1,
                        float* const GPU_gw_average_color_2,
                        float* const GPU_gw_auto_correlation_2,
                        float* const GPU_gw_cross_correlation_3,
                        float* const GPU_gw_normalized_score_4,
                        int* match_matrix,
                        int* disparity_map,
                        int* depth_map,
                        const int numRows, const int numCols,
                        const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(32, 32);
  //const dim3 blockSize(8,8);
  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  //const dim3 gridSize(15,12);
  //const dim3 gridSize(5,1);
  //const dim3 gridSize(1,1);
  const dim3 gridSize(ceilf(static_cast<float>(numCols) / blockSize.x), ceilf(static_cast<float>(numRows) / blockSize.y));

  //TODO: Launch a kernel for separating the RGBA image into different color channels
  //separateChannels<<<gridSize,blockSize>>>(d_inputImageRGBA,(int)numRows,(int)numCols,d_red, d_green,d_blue);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  //hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //TODO: Call your convolution kernel here 3 times, once for each color channel.
  int mid_shared_address = (blockSize.x + (2 * (filterWidth / 2))) * (blockSize.y + (2 * (filterWidth / 2))) * sizeof(unsigned char);
  Kernel_1 <<< gridSize, blockSize, (blockSize.x + (2 * (filterWidth / 2)))*(blockSize.y + (2 * (filterWidth / 2)))*sizeof(unsigned char) * 2 >>> (d_inputImageGray_1,
      d_inputImageGray_2,
      d_outputImageGray,
      GPU_gw_average_color_1,
      GPU_gw_average_color_2,
      GPU_gw_auto_correlation_1,
      GPU_gw_auto_correlation_2,
      (int)numRows,
      (int)numCols,
      d_filter,
      filterWidth, mid_shared_address);
  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  //for (int r = 0; r < (int)numRows; ++r) {
  //    for (int c = 0; c < (int)numCols; ++c) {
  //     printf("%f ",GPU_gw_auto_correlation_1[r * numCols + c]);
  //    }
  //    printf("\n");
  //}


  Kernel_2 <<< gridSize, blockSize, (blockSize.x + (2 * (filterWidth / 2)))*(blockSize.y + (2 * (filterWidth / 2)))*sizeof(char) * 2 >>> (d_inputImageGray_1,
      d_inputImageGray_2,
      d_outputImageGray,
      GPU_gw_average_color_1,
      GPU_gw_average_color_2,
      GPU_gw_auto_correlation_1,
      GPU_gw_auto_correlation_2,
      GPU_gw_cross_correlation_3,
      GPU_gw_normalized_score_4,
      match_matrix,
      disparity_map,
      depth_map,
      (int)numRows,
      (int)numCols,
      d_filter,
      filterWidth, mid_shared_address);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  /*


   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

   // Now we recombine your results. We take care of launching this kernel for you.
   //
   // NOTE: This kernel launch depends on the gridSize and blockSize variables,
   // which you must set yourself.
   recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                              d_greenBlurred,
                                              d_blueBlurred,
                                              d_outputImageRGBA,
                                              numRows,
                                              numCols);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


   recombineChannels<<<gridSize, blockSize>>>(d_redBlurred_1,
                                              d_greenBlurred_1,
                                              d_blueBlurred_1,
                                              d_outputImageRGBA_1,
                                              numRows,
                                              numCols);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  */

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated

/*
void cleanup() {
  checkCudaErrors(hipFree(d_inputImageGray_1));//d_red));
  checkCudaErrors(hipFree(d_inputImageGray_2));//d_green));
  checkCudaErrors(hipFree(d_outputImageGray));//d_blue));
}
*/