#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2012-, Open Perception, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of the copyright holder(s) nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *
 */

/** \brief Adaptive Cost 2-pass Scanline Optimization Stereo Matching algorithm implementation
  * please see related documentation on stereo/stereo_matching.h
  *
  * \author Aras Dargazany
  * \ingroup stereo matching
  */

//#include "projects/icarus/sensor_processing/libstereo/tests/stereo_matching.h"
#include "projects/stereo_traversability_experiments/aras/libstereo/tTestStereoMatching.h"

#include "projects/stereo_traversability_experiments/aras/libstereo/gpu/tDeviceWrapper.hpp"
#include "projects/stereo_traversability_experiments/aras/libstereo/tTimer.hpp"


#include <stdio.h> //stderr - fprintf
//#include <iostream> //cout - cerr
//#include <iomanip> //setprecision


const int radius_ = 5;
//const int smoothness_strong_ = 100;
//const int smoothness_weak_ = 20;
const int max_disp_ = 60;
const int x_off_ = 0;
const float numeric_limits_float_max = 1e+20; //std::numeric_limits<float>::max();

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void compute_parallel(const unsigned char* ref_img, const unsigned char* trg_img,
                                 const unsigned int height_, const unsigned int width_,
                                 const float *ds, const float* lut,
                                 float* wl, float* acc,
                                 float *fwd,
                                 float *bck,
                                 short int* disp_map_)

{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x; //rows -- y -- height

  // except for special cases, the total number of threads in all blocks
  // adds up to more than the vector length n, so this conditional is
  // EXTREMELY important to avoid writing past the allocated memory for
  // the vector y.
  /*
  * main loop to iterate through the entire left and right images
  */


  shared int* array;
  uint y_min = radius_ + 1;
  uint y_max = height_ - radius_;
  uint y = blockIdx.x + y_min;

  /*! scanning Y - row - height of the image*/
  if (y < y_max)
  {
    /*! 1st scan X - accumulator for SAD*/
    uint x_min = x_off_ + max_disp_ + 1;
    uint x_max = width_;
    uint x = threadIdx.x + x_min;

//    float wl[2 * radius_ + 1]; //  float *wl = new float [ 2 * radius_ + 1 ];
////    hipMalloc((void**)&wl, (2 * radius_ + 1)* sizeof(float));
//    float acc[686 * max_disp_]; //width_ = 686
////    hipMalloc((void**)&acc, width_ * max_disp_ * sizeof(float));
////    hipMemset(acc, 0, width_ * max_disp_ * sizeof(float));


    if (x < x_max)
    {

      for (int j = -radius_; j <= radius_; j++)
      {
        wl[j + radius_] = lut[ abs(ref_img[(y + j) * width_ + x] - ref_img[y * width_ + x]) ] * ds[j + radius_];
      }// j

      for (int d = 0; d < max_disp_; d++)
      {
        float sumw  = 0.0;
        float num = 0.0;

        for (int j = -radius_; j <= radius_; j++)
        {
          float weight_r = lut[ abs(trg_img[(y + j) * width_ + x - d - x_off_] - trg_img[y * width_ + x - d - x_off_]) ] * ds[j + radius_];
          int sad = abs(ref_img[(y + j) * width_ + x] - trg_img[(y + j) * width_ + x - d - x_off_]);
          num += wl[j + radius_] * weight_r * static_cast<float>(sad);
          sumw += wl[j + radius_] * weight_r;
        }

        //acc[x][d] = num / sumw;
        acc[(x * max_disp_) + d] = num / sumw;
      }//d

    }//x

    /*! 4th scan X - last scan - to fill disp_map_*/
    if (x < x_max)
    {
      float c_min = numeric_limits_float_max;
      short int dbest = 0;

      for (int d = 0; d < max_disp_; d++)
      {
        //acc[(x * max_disp_) + d] = fwd[(x * max_disp_) + d] + bck[(x * max_disp_) + d];
        if (acc[(x * max_disp_) + d] < c_min)
        {
          c_min = acc[(x * max_disp_) + d];
          dbest = static_cast<short int>(d);
        }
      }

      disp_map_[(y * width_) + x] = static_cast<short int>(dbest * 16);

    } //x last scan for disp_map_


  }// y_iter - row - height of img

}// end kernel

//////////////////////////////////////////////////////////////////////////////
void
finroc::stereo_traversability_experiments::aras::libstereo::tTestACSO::compute_impl_cuda(unsigned char* ref_img, unsigned char* trg_img)
{

  /////////////////////////////////////
  // some variables and configuration section
  /////////////////////////////////////
  const unsigned long N = width_ * height_; //123456;
//  std::cout << "width_: " << width_ << std::endl;
//  static int numThreadsPerBlock = 256;
  static int selectedDevice = 0;   // device to use in case there is more than one
  //std::cout << "numeric_limits_float_max: " << numeric_limits_float_max << std::endl;



  /////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    fprintf(stderr, "Sorry, no CUDA device fount");
    //return 1;
  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount - 1);
    //return 1;
  }
  hipSetDevice(selectedDevice);
  checkErrors("initialisations");

  /////////////////////////////////////
  // (2) initialise data on the CPU
  /////////////////////////////////////
  /*
   * initializaing or constructing the variables
   */
  float *wl_gpu = new float [ 2 * radius_ + 1 ];
  hipMalloc((void**)&wl_gpu, (2 * radius_ + 1)* sizeof(float));
  float* acc_gpu; //width_ = 686
  hipMalloc((void**)&acc_gpu, width_ * max_disp_ * sizeof(float));
  hipMemset(acc_gpu, 0, width_ * max_disp_ * sizeof(float));

  //spatial distance init
  float *ds = new float[ 2 * radius_ + 1 ];
  for (int j = -radius_; j <= radius_; j++)
    ds[j + radius_] = static_cast<float>(exp(- abs(j) / gamma_s_));

  //LUT for color distance weight computation
  float lut[256];
  for (int j = 0; j < 256; j++)
    lut[j] = float(exp(-j / gamma_c_));

  /////////////////////////////////////
  // (3) allocate memory on host (main CPU memory) and device,
  //     h_ denotes data residing on the host, d_ on device
  /////////////////////////////////////
  unsigned char *ref_img_gpu;
  hipMalloc((void**)&ref_img_gpu, height_ * width_ * sizeof(unsigned char));
  unsigned char *trg_img_gpu;
  hipMalloc((void**)&trg_img_gpu, height_ * width_ * sizeof(unsigned char));
  float* ds_gpu;
  hipMalloc((void**)&ds_gpu, (2 * radius_ + 1) * sizeof(float));
  float* lut_gpu;

  hipMalloc((void**)&lut_gpu, 256 * sizeof(float));
  float* fwd_gpu;
  hipMalloc((void**)&fwd_gpu, width_ * max_disp_ * sizeof(float));
  hipMemset(fwd_gpu, 0, width_ * max_disp_ * sizeof(float));
  float* bck_gpu;
  hipMalloc((void**)&bck_gpu, width_ * max_disp_ * sizeof(float));
  hipMemset(bck_gpu, 0, width_ * max_disp_ * sizeof(float));

  /*! final output disparity map*/
  short int* disp_map_gpu_;
  hipMalloc((void**)&disp_map_gpu_, height_ * width_ * sizeof(short int));
  hipMemset(disp_map_gpu_, 0, sizeof(short int)*height_ * width_);

  checkErrors("memory allocation");


  /////////////////////////////////////
  // (4) copy data to device
  /////////////////////////////////////
  hipMemcpy(trg_img_gpu, trg_img, N * sizeof(uchar), hipMemcpyHostToDevice);
  hipMemcpy(lut_gpu,     lut, 256 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ds_gpu, ds, (2 * radius_ + 1) * sizeof(float), hipMemcpyHostToDevice);
  checkErrors("copy data to device");

  /////////////////////////////////////
  // (6) perform computation on device
  //     - we use numThreadsPerBlock threads per block
  //     - the total number of blocks is obtained by rounding the
  //       vector length N up to the next multiple of numThreadsPerBlock
  /////////////////////////////////////

  /*/
  // device kernel setup
  /*/
  //dim3 block(256,1,1);
//  dim3 block(1024, 1, 1);

  uint x_min = x_off_ + max_disp_ + 1;
  uint x_max = width_;
  uint x_length = x_max - x_min;
  dim3 block(x_length, 1, 1);

  //uint x_length = x_max - x_min;
  //dim3 grid((unsigned)ceil((N) / (double)(block.x)), 1, 1);
//  dim3 grid((unsigned)ceil((y_length) / (double)(block.x)), 1, 1);
//  dim3 grid(1, 1, 1);
  uint y_min = radius_ + 1;
  uint y_max = height_ - radius_;
  uint y_length = y_max - y_min;
  dim3 grid(y_length, 1, 1);


  //dim3 grid(65535, 1, 1);  //67107840
  std::cout << "Launch config: block(";
  std::cout << block.x << "x" << block.y << "x" << block.z << "), grid(";
  std::cout << grid.x << "x" << grid.y << "x" << grid.z << ")" << std::endl;
  //int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

  /*/
  // run kernel
  /*/
  hipDeviceSynchronize();
  Timer timer;
  timer.start();
//  compute_parallel <<< grid, block>>>(ref_img_gpu, trg_img_gpu, height_, width_,
//                                      ds_gpu, lut_gpu,
//                                      fwd_gpu,
//                                      bck_gpu,
//                                      disp_map_gpu_);

//  compute_parallel <<<grid, block>>>(ref_img_gpu, trg_img_gpu, height_, width_,
//                                     ds_gpu, lut_gpu,
//                                     wl_gpu, acc_gpu,
//                                     fwd_gpu,
//                                     bck_gpu,
//                                     disp_map_gpu_);
  hipDeviceSynchronize();
  double cudatime = timer.stop();
  checkErrors("compute on device");

  /*/
  // print out timinigs
  /*/
  double throughput = 3.0 * N * sizeof(double) / (cudatime * 1e9);
  std::cout << "CUDA implementation     : ";
  std::cout << "Time: " << std::fixed << std::setprecision(4) << cudatime << "s";
  std::cout << ", throughput: " << std::setprecision(2) << throughput << " GB/s";
  std::cout << std::endl;


  /////////////////////////////////////
  // (7) read back result from device into temp vector
  /////////////////////////////////////
  hipMemcpy(disp_map_, disp_map_gpu_, N * sizeof(short int), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");

  /////////////////////////////////////
  // (9) clean up, free memory on gpu
  /////////////////////////////////////
  hipFree(ref_img_gpu);
  hipFree(trg_img_gpu);
  hipFree(disp_map_gpu_);
  hipFree(ds_gpu);
  hipFree(lut_gpu);
  hipFree(fwd_gpu);
  hipFree(bck_gpu);
  hipFree(wl_gpu);
  hipFree(acc_gpu);


  /*
   * deconstructing the variables on cpu
   */
  delete [] ds;

} //compute_impl_cuda
